
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <string.h>
#include <time.h>
#include <stdlib.h>
#include <math.h>
#include <limits.h>
#include <omp.h>

int N;                // Number of vertices
int **adj_matrix;     // Adjacency matrix representation of the graph

typedef struct LINKED_LIST {
    int v;
    struct LINKED_LIST *parent;
} linked_list;

void generate_combinations(int level, int ***combinations, linked_list *x, int subset_size, int original_subset_size) {
    // Calculate the cost for combination when reached the end of it
    if (level == 0) {
        // Create a subset array to store the vertices in the combination
        int *subset = (int *)malloc(original_subset_size * sizeof(int));
        int iter = 0;
        while (x) {
            if (x->v > 0) {
                subset[iter] = x->v;
                iter++;
            }
            x = x->parent;
        }
        int bits = 0;
        for (int i = 0; i < original_subset_size; i++) {
            bits |= 1 << subset[i];
        }

        // Calculate the cost for each vertex in the combination and store the minimum cost and its parent
        for (int i = 0; i < original_subset_size; i++) {
            int prev = bits & ~(1 << subset[i]);
            int cost_list[2] = {INT_MAX, INT_MAX};
            for (int j = 0; j < original_subset_size; j++) {
                int cost = combinations[prev][subset[j]][0] + adj_matrix[subset[j]][subset[i]];
                if (subset[j] != subset[i] && cost < cost_list[0]) {
                    cost_list[0] = cost;
                    cost_list[1] = subset[j];
                }
            }
            // Store the minimum cost and its parent in the combinations array
            memcpy(combinations[bits][subset[i]], cost_list, 2 * sizeof(int));
        }
        free(subset);
    } else {
        linked_list l1, l2;
        // Generate combinations by considering two possibilities:
        // 1. Exclude the current level vertex
        // 2. Include the current level vertex

        if (level > subset_size) {
            // Exclude the current level vertex (set it to 0) and continue generating combinations
            l1.v = 0;
            l1.parent = x;
            generate_combinations(level - 1, combinations, &l1, subset_size, original_subset_size);
        }
        if (subset_size > 0) {
            // Include the current level vertex and continue generating combinations with a reduced subset size
            l2.v = level;
            l2.parent = x;
            generate_combinations(level - 1, combinations, &l2, subset_size - 1, original_subset_size);
        }
    }
}


// TSP function to calculate the minimum cost path
int tsp(int *path) {
    int ***combinations;
    int combinations_d1 = pow(2, N) - 1;
    combinations = (int ***)malloc(combinations_d1 * N * 2 * sizeof(int));
    for (int i = 0; i < combinations_d1; i++) {
        combinations[i] = (int **)malloc(N * 2 * sizeof(int));
        for (int j = 0; j < N; j++) {
            combinations[i][j] = (int *)malloc(2 * sizeof(int));
        }
    }

    // Initialize combinations for single vertices
    for (int i = 1; i < N; i++) {
        combinations[1 << i][i][0] = adj_matrix[0][i];
        combinations[1 << i][i][1] = 0;
    }

    // Generate combinations for subset sizes greater than 1
    for (int subset_size = 2; subset_size < N; subset_size++) {
        generate_combinations(N - 1, combinations, NULL, subset_size, subset_size);
    }

    // Calculate optimal cost
    int bits = (pow(2, N) - 1) - 1;
    int optimum = INT_MAX;
    int parent;
    for (int i = 1; i < N; i++) {
        int cost = combinations[bits][i][0] + adj_matrix[i][0];
        if (cost < optimum) {
            optimum = cost;
            parent = i;
        }
    }

    // Retrieve the optimal path
    for (int i = N - 1; i > 0; i--) {
        path[i] = parent;
        int tmp = bits & ~(1 << parent);
        parent = combinations[bits][parent][1];
        bits = tmp;
    }

    path[0] = 0;

    // Freeing the memory
    for (int i = 0; i < combinations_d1; i++) {
        for (int j = 0; j < N; j++) {
            free(combinations[i][j]);
        }
        free(combinations[i]);
    }
    free(combinations);

    return optimum;
}

int main(int argc, char *argv[]) {
    if (argc < 2) {
        printf("No instance filename given\n");
        return -1;
    }

    // Loading size and adjacency matrix from file
    FILE *instance_file;
    instance_file = fopen(argv[1], "r");
    if (instance_file == NULL) {
        printf("Error reading the file\n");
        return -1;
    }
    fscanf(instance_file, "%d", &N);
    adj_matrix = (int **)malloc(N * N * sizeof(int));
    for (int i = 0; i < N; i++) {
        adj_matrix[i] = (int *)malloc(N * sizeof(int));
    }
    for (int i = 0; i < N; ++i) {
        for (int j = 0; j < N; ++j) {
            fscanf(instance_file, "%d", adj_matrix[i] + j);
        }
    }
    fclose(instance_file);

    // Running tsp and measuring time
    int out_path[N];
    clock_t start, end;
    double cpu_time_used;
    start = clock();
    int min_cost = tsp(out_path);
    end = clock();
    cpu_time_used = ((double)(end - start)) / CLOCKS_PER_SEC;

    // Printing output
    printf("Time: %f\n", cpu_time_used);
    printf("Path: ");
    for (int i = 0; i < N; i++)
        printf("%d ", out_path[i]);
    printf("\n");
    printf("Minimum cost: %d\n", min_cost);

    // Cleaning resources
    for (int i = 0; i < N; i++) {
        free(adj_matrix[i]);
    }
    free(adj_matrix);

    return 0;
}
