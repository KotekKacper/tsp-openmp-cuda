#include <stdio.h>
#include <string.h>
#include <time.h>
#include <stdlib.h>
#include <math.h>
#include <limits.h>
#include <omp.h>
#include <hip/hip_runtime.h>

int N;
int **adj_matrix;
int *adj_matrix_flat;

typedef struct LINKED_LIST {
    int v;
    struct LINKED_LIST * parent;
} linked_list;

__device__ int global_adj_matrix_flat[500]; 

// Kernel function to save the global adjacency matrix flat on the GPU
__global__ void save_global_adj_matrix_flat(int* input_data, int data_size) {
    int thread_id = blockIdx.x * blockDim.x + threadIdx.x;
    if (thread_id < data_size) {
        global_adj_matrix_flat[thread_id] = input_data[thread_id];
    }
}

// Kernel function to calculate the cost using CUDA
__global__ void calculate_cost_cuda(int* subset, int subset_size, int* cost_list, int k, int* combinations, int N) {
    int thread_id = threadIdx.x;
    if (thread_id < subset_size && thread_id != k) {
        int cost = combinations[subset[thread_id]] + global_adj_matrix_flat[subset[thread_id] * N + subset[k]];
        atomicMin(&cost_list[0], cost);
        if (cost == cost_list[0]) {
            cost_list[1] = subset[thread_id];
        }
    }
}
     
void generate_combinations(int level, int ***combinations, linked_list * x, int subset_size, int original_subset_size) {
    // Calculate the cost for combination when reached the end of it
    if (level==0) {
        // Create a subset array to store the vertices in the combination
        int *subset = (int *)malloc(original_subset_size * sizeof(int));
        int iter = 0;
        while (x) { 
            if(x->v > 0){
                subset[iter] = x->v;
                iter++;
            }
            x = x -> parent;
        }
        int bits = 0;
        for(int i = 0; i < original_subset_size; i++) {
            bits |= 1 << subset[i];
        }

        // Calculate the cost for each vertex in the combination and store the minimum cost and its parent
        #pragma omp parallel for
        for(int i=0; i<original_subset_size; i++) {
            int prev = bits & ~(1 << subset[i]);
            int cost_list[2] = {INT_MAX,0};

            // Paralization with CUDA
            // Alocating GPU memory
            int* d_subset;
            int* d_cost_list;
            int* d_combinations;
            hipMalloc((void**)&d_subset, original_subset_size * sizeof(int));
            hipMalloc((void**)&d_cost_list, 2 * sizeof(int));
            hipMalloc((void**)&d_combinations, N * sizeof(int));
            // Copying data to GPU memory
            hipMemcpy(d_subset, subset, original_subset_size * sizeof(int), hipMemcpyHostToDevice);
            hipMemcpy(d_cost_list, cost_list, 2 * sizeof(int), hipMemcpyHostToDevice);
            hipMemcpy(d_combinations, combinations[prev][0], N * sizeof(int), hipMemcpyHostToDevice);
            // Running calculate_cost_cuda on GPU
            int threadsPerBlock = original_subset_size;
            int gridSize = 1;
            calculate_cost_cuda<<<gridSize, threadsPerBlock>>>(d_subset, original_subset_size, d_cost_list, i, d_combinations, N);
            // Loading data back from GPU
            hipMemcpy(cost_list, d_cost_list, 2 * sizeof(int), hipMemcpyDeviceToHost);
            // Releasing GPU memory
            hipFree(d_subset);
            hipFree(d_cost_list);
            hipFree(d_combinations);

            // Store the minimum cost and its parent in the combinations array
            combinations[bits][0][subset[i]] = cost_list[0];
            combinations[bits][1][subset[i]] = cost_list[1];
        }
        free(subset);
    } else {
        linked_list l1, l2;
        // Generate combinations by considering two possibilities:
        // 1. Exclude the current level vertex
        // 2. Include the current level vertex

        #pragma omp task
        {
        if (level > subset_size) {
            // Exclude the current level vertex (set it to 0) and continue generating combinations
            l1.v = 0;
            l1.parent = x;
            generate_combinations(level - 1, combinations, &l1, subset_size, original_subset_size);
        }
        }
        #pragma omp task
        {
            if (subset_size > 0) {
                // Include the current level vertex and continue generating combinations with a reduced subset size
                l2.v = level;
                l2.parent = x;
                generate_combinations(level - 1, combinations, &l2, subset_size - 1, original_subset_size);
            }
        }
        #pragma omp taskwait  
    }
}

// TSP function to calculate the minimum cost path
int tsp(int *path) {
    int ***combinations;
    int combinations_d1 = pow(2, N) - 1;
    combinations = (int ***)malloc(combinations_d1 * N * 2 * sizeof(int));
    for(int i=0; i<combinations_d1;i++) {
        combinations[i] = (int **)malloc(N * 2 * sizeof(int));
        for(int j=0; j<2; j++) {
            combinations[i][j] = (int *)malloc(N * sizeof(int));
        }
    }

    // Initialize combinations for single vertices
    for(int i=1; i<N; i++) {
        combinations[1<<i][0][i] = adj_matrix_flat[0*N+i];
        combinations[1<<i][1][i] = 0;
    }

    // Generate combinations for subset sizes greater than 1
    for(int subset_size=2; subset_size<N; subset_size++){
        generate_combinations(N-1, combinations, NULL, subset_size, subset_size);
    }

    // Calculate optimal cost
    int bits = (pow(2, N) - 1) - 1;
    int optimum = INT_MAX;
    int parent;
    for (int i = 1; i < N; i++) {
        int cost = combinations[bits][0][i] + adj_matrix[i][0];
        if (cost < optimum) {
            optimum = cost;
            parent = i;
        }
    }

    // Retrieve the optimal path
    for (int i = N - 1; i > 0; i--) {
        path[i] = parent;
        int tmp = bits & ~(1 << parent);
        parent = combinations[bits][1][parent];
        bits = tmp;
    }

    path[0] = 0;

    // Freeing the memory
    free(combinations);

    return optimum;
}

int main(int argc, char *argv[]) {
    if (argc < 2) {
        printf("No instance filename given\n");
        return -1;
    }

    // Loading size and adjacency matrix from file
    FILE *instance_file;
    instance_file = fopen(argv[1], "r");
    if (instance_file == NULL) {
        printf("Error reading the file\n");
        return -1;
    }
    fscanf(instance_file, "%d", &N);
    adj_matrix = (int **)malloc(N * N * sizeof(int));
    for (int i = 0; i < N; i++) {
        adj_matrix[i] = (int *)malloc(N * sizeof(int));
    }
    for (int i = 0; i < N; ++i) {
        for (int j = 0; j < N; ++j) {
            fscanf(instance_file, "%d", adj_matrix[i] + j);
        }
    }
    fclose(instance_file);

    adj_matrix_flat = (int *)malloc(N * N * sizeof(int));
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            adj_matrix_flat[i * N + j] = adj_matrix[i][j];
        }
    }
    
    // Copying adj_matrix_flat to CUDA
    int data_size = N*N;
    int* d_input_data;
    hipMalloc((void**)&d_input_data, data_size * sizeof(int));
    hipMemcpy(d_input_data, adj_matrix_flat, data_size * sizeof(int), hipMemcpyHostToDevice);
    int threadsPerBlock = 128;
    int gridSize = (threadsPerBlock + data_size - 1) / threadsPerBlock;
    save_global_adj_matrix_flat<<<gridSize, threadsPerBlock>>>(d_input_data, data_size);

    // Running tsp and measuring time
    int out_path[N];
    clock_t start, end;
    double cpu_time_used;
    start = clock();
    int min_cost = tsp(out_path);
    end = clock();
    cpu_time_used = ((double)(end - start)) / CLOCKS_PER_SEC;

    // Printing output
    printf("Time: %f\n", cpu_time_used);
    printf("Path: ");
    for (int i = 0; i < N; i++)
        printf("%d ", out_path[i]);
    printf("\n");
    printf("Minimum cost: %d\n", min_cost);

    // Cleaning resources
    for (int i = 0; i < N; i++) {
        free(adj_matrix[i]);
    }
    free(adj_matrix);
    free(adj_matrix_flat);

    return 0;
}